#include "hip/hip_runtime.h"
/*
Copyright (C) 2020 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <config.h>
#include <iostream>

#include "cuda_error.h"
#include "node_group.h"
#include "neurongpu.h"

__constant__ NodeGroupStruct NodeGroupArray[MAX_N_NODE_GROUPS];
__device__ signed char *NodeGroupMap;

__global__
void NodeGroupMapInit(signed char *node_group_map)
{
  NodeGroupMap = node_group_map;
}

int NeuronGPU::NodeGroupArrayInit()
{
  gpuErrchk(hipMalloc(&d_node_group_map_,
		       node_group_map_.size()*sizeof(signed char)));

  std::vector<NodeGroupStruct> ngs_vect;
  for (unsigned int i=0; i<node_vect_.size(); i++) {
    NodeGroupStruct ngs;
    ngs.node_type_ = node_vect_[i]->node_type_;
    ngs.i_node_0_ = node_vect_[i]->i_node_0_;
    ngs.n_node_ = node_vect_[i]->n_node_;
    ngs.n_port_ = node_vect_[i]->n_port_;
    ngs.n_param_ = node_vect_[i]->n_param_;
    ngs.get_spike_array_ = node_vect_[i]->get_spike_array_;
    ngs.intern_get_spike_array_ = node_vect_[i]->intern_get_spike_array_;

    ngs.spike_count_ = node_vect_[i]->spike_count_;
    ngs.rec_spike_times_ = node_vect_[i]->rec_spike_times_;
    ngs.n_rec_spike_times_ = node_vect_[i]->n_rec_spike_times_;
    ngs.max_n_rec_spike_times_ = node_vect_[i]->max_n_rec_spike_times_;
    ngs.den_delay_arr_ = node_vect_[i]->den_delay_arr_;
    
    ngs_vect.push_back(ngs);
  }
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(NodeGroupArray), ngs_vect.data(),
			       ngs_vect.size()*sizeof(NodeGroupStruct)));

  gpuErrchk(hipMemcpy(d_node_group_map_, node_group_map_.data(),
		       node_group_map_.size()*sizeof(signed char),
		       hipMemcpyHostToDevice));
  NodeGroupMapInit<<<1, 1>>>(d_node_group_map_);

  return 0;
}

double *NeuronGPU::InitGetSpikeArray (int n_node, int n_port)
{
  double *d_get_spike_array = NULL;
  if (n_node*n_port > 0) {
    gpuErrchk(hipMalloc(&d_get_spike_array, n_node*n_port
			 *sizeof(double)));
  }
  
  return d_get_spike_array;
}

int NeuronGPU::FreeNodeGroupMap()
{
  gpuErrchk(hipFree(d_node_group_map_));
	    
  return 0;
}
